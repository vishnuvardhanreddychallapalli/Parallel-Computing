
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

 inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
     if (code != hipSuccess) 
   {
       // cudaGetErrorString is an api that will pop out an error to user screen else all errors in gpu are silent....
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// All gpu function start with __global__ called from cpu...
__global__ void polynomial_expansion (float* poly, int degree,
               int n, float* array) {
  //TODO: Write code to use the GPU here!
  //code should write the output back to array
   int curr_index = (blockIdx.x * blockDim.x )+ threadIdx.x;
    if( curr_index < n )
    {
      float out = 0.;
      float xtothepowerof = 1.;
      for (int i=0; i<=degree; ++i) {
        out += xtothepowerof*poly[i];
        xtothepowerof *=array[curr_index];
      }
      array[curr_index]=out;
    }
}

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }
  
  int n = atoi(argv[1]); //TODO: atoi is an unsafe function....
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;
  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;
  float* d_array;
  float* d_poly;
  /// Giving pointers for gpu..
  gpuErrchk(hipMalloc(&d_poly, (degree+1)* sizeof(float)));
  gpuErrchk(hipMalloc(&d_array, n * sizeof(float)));
  gpuErrchk(hipMemcpy(d_array, array, (n) * sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_poly, poly, (degree+1)*sizeof(float),hipMemcpyHostToDevice));
   std::chrono::time_point<std::chrono::system_clock> begin, end;
   begin = std::chrono::system_clock::now();
   int block_size=256;
   int no_of_blocks=(n+block_size-1)/block_size;
   polynomial_expansion<<<no_of_blocks,block_size>>>(d_poly, degree, n, d_array);
   /// Copying the final_array to input_array...
  gpuErrchk( hipMemcpy( array, d_array, n * sizeof(float),hipMemcpyDeviceToHost)); 
  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
    ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
   hipFree(d_array);
   hipFree(d_poly);
   hipDeviceSynchronize();
   end = std::chrono::system_clock::now();
   std::chrono::duration<double> totaltime = (end-begin)/nbiter;
   std::cerr<<array[0]<<std::endl;
   std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

   delete[] array;
   delete[] poly;

  return 0;
}